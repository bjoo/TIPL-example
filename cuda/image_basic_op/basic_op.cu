#include "TIPL/tipl.hpp"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


int main(void)
{
    // 1: load atlas (integers representing brain regions)
    tipl::image<3> hfrom;
    if(!hfrom.load_from_file<tipl::io::nifti>("100206_T1w.nii"))
    {
        std::cout << "cannot find the sample file" << std::endl;
        return 1;
    }
    // enlarge it
    tipl::upsampling(hfrom);
    tipl::upsampling(hfrom);

    tipl::image<3> hto(hfrom.shape());
    tipl::device_image<3> dto(hfrom.shape()),dfrom(hfrom);


    // use single thread
    {
        tipl::time t("single thread time:");

        for(size_t i = 0;i < hfrom.size();++i)
            if(hfrom[i] > 0)
                hto[i] = hfrom[i]*5.5f+100.0f;

    }
    // use multi thread
    {
        tipl::time t("par_for multi-thread time:");

        tipl::par_for(hfrom.size(),[&](size_t i)
        {
            if(hfrom[i] > 0)
                hto[i] = hfrom[i]*5.5f+100.0f;
        });

    }
    // use cuda
    {
        auto from = tipl::make_shared(dfrom);
        auto to = tipl::make_shared(dto);
        tipl::time t("cuda_for time:");

        tipl::cuda_for(from.size(),[=]__device__(size_t i) mutable
        {
            if(from[i] > 0)
               to[i] = from[i]*5.5f+100.0f;
        });
        std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
    }

    return 0;
}
