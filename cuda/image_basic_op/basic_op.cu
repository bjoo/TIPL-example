#include "tipl/tipl.hpp"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


int main(void)
{
    // 1: load atlas (integers representing brain regions)
    tipl::image<3> hfrom;
    if(!hfrom.load_from_file<tipl::io::nifti>("100206_T1w.nii"))
    {
        std::cout << "cannot find the sample file" << std::endl;
        return 1;
    }
    // enlarge it
    tipl::upsampling(hfrom);
    tipl::upsampling(hfrom);

    tipl::image<3> hto(hfrom.shape());
    tipl::device_image<3> dto(hfrom.shape()),dfrom(hfrom);


    // use single thread
    {
        tipl::time t("single thread time:");

        for(size_t i = 0;i < hfrom.size();++i)
            if(hfrom[i] > 0)
                hto[i] = hfrom[i]*5.5f+100.0f;

    }
    // use multi thread
    {
        tipl::time t("simple multithread time:");

        (hto = hfrom[hfrom > 0]*5.5f+100.0f)
                >> tipl::backend::mt();

    }
    // use multi thread
    {
        tipl::time t("par_for multi-thread time:");

        tipl::par_for(hfrom.size(),[&](size_t i)
        {
            if(hfrom[i] > 0)
                hto[i] = hfrom[i]*5.5f+100.0f;
        });

    }
    // use cuda
    {
        auto from = tipl::make_alias(dfrom);
        auto to = tipl::make_alias(dto);
        tipl::time t("cuda_for time:");

        tipl::cuda_for(from.size(),[=]__device__(size_t i)
        {
           if(from[i] > 0)
               to[i] = from[i]*5.5f+100.0f;
        });
        std::cout << hipGetErrorName(hipGetLastError()) << std::endl;
    }

    return 0;
}
